#include "hip/hip_runtime.h"
#include "cuspCUDA.cuh"
#include "cuspCUDA.hpp"

namespace cusp {

template <typename TElem, typename TInt>
__global__ void matrix_element_add_kernel(TElem *a, TElem b, TInt row, TInt col)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (blockIdx.x < row && threadIdx.x < col)
		a[i] += b;
}

template <typename TElem, typename TInt>
__global__ void matrix_element_sub_kernel(TElem *a, TElem b, TInt row, TInt col)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (blockIdx.x < row && threadIdx.x < col)
		a[i] -= b;
}

template <typename TElem, typename TInt>
__global__ void matrix_element_mult_kernel(TElem *a, TElem b, TInt row, TInt col)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (blockIdx.x < row && threadIdx.x < col)
		a[i] *= b;
}

template <typename TElem, typename TInt>
__global__ void matrix_element_div_kernel(TElem *a, TElem b, TInt row, TInt col)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (blockIdx.x < row && threadIdx.x < col)
		a[i] /= b;
}

template <typename TElem, typename TInt>
__global__ void matrix_matrix_add_kernel(TElem *a, TElem *b, TInt row, TInt col)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (blockIdx.x < row && threadIdx.x < col)
		a[i] += b[i];
}

template <typename TElem, typename TInt>
__global__ void matrix_matrix_sub_kernel(TElem *a, TElem *b, TInt row, TInt col)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (blockIdx.x < row && threadIdx.x < col)
		a[i] -= b[i];
}

template <typename TElem, typename TInt>
void matrix_element_add(TElem *a, TElem b, TInt row, TInt col)
{
	matrix_element_add_kernel<TElem, TInt> <<<row, col>>>(a, b, row, col);
}

template <typename TElem, typename TInt>
void matrix_element_sub(TElem *a, TElem b, TInt row, TInt col)
{
	matrix_element_sub_kernel<TElem, TInt> <<<row, col>>>(a, b, row, col);
}

template <typename TElem, typename TInt>
void matrix_element_mult(TElem *a, TElem b, TInt row, TInt col)
{
	matrix_element_mult_kernel<TElem, TInt> <<<row, col>>>(a, b, row, col);
}

template <typename TElem, typename TInt>
void matrix_element_div(TElem *a, TElem b, TInt row, TInt col)
{
	matrix_element_div_kernel<TElem, TInt> <<<row, col>>>(a, b, row, col);
}

template <typename TElem, typename TInt>
void matrix_matrix_add(TElem *a, TElem *b, TInt row, TInt col)
{
	matrix_matrix_add_kernel<TElem, TInt> <<<row, col>>>(a, b, row, col);
}

template <typename TElem, typename TInt>
void matrix_matrix_sub(TElem *a, TElem *b, TInt row, TInt col)
{
	matrix_matrix_sub_kernel<TElem, TInt> <<<row, col>>>(a, b, row, col);
}

}

#include "cuspCUDA.tpp"
