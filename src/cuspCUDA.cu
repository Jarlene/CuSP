#include "hip/hip_runtime.h"
#include "cuspCUDA.cuh"
#include "cuspCUDA.hpp"

namespace cusp {

template <typename TElem, typename TInt>
__global__ void matrix_element_add_kernel(TElem *a, TElem b, TInt nRow, TInt nCol)
{
	if (blockIdx.x < nRow && threadIdx.x < nCol) {
		TInt i = blockDim.x * blockIdx.x + threadIdx.x;
		a[i] += b;
	}
}

template <typename TElem, typename TInt>
__global__ void matrix_element_sub_kernel(TElem *a, TElem b, TInt nRow, TInt nCol)
{
	if (blockIdx.x < nRow && threadIdx.x < nCol) {
		TInt i = blockDim.x * blockIdx.x + threadIdx.x;
		a[i] -= b;
	}
}

template <typename TElem, typename TInt>
__global__ void matrix_element_mult_kernel(TElem *a, TElem b, TInt nRow, TInt nCol)
{
	if (blockIdx.x < nRow && threadIdx.x < nCol) {
		TInt i = blockDim.x * blockIdx.x + threadIdx.x;
		a[i] *= b;
	}
}

template <typename TElem, typename TInt>
__global__ void matrix_element_div_kernel(TElem *a, TElem b, TInt nRow, TInt nCol)
{
	if (blockIdx.x < nRow && threadIdx.x < nCol) {
		TInt i = blockDim.x * blockIdx.x + threadIdx.x;
		a[i] /= b;
	}
}

template <typename TElem, typename TInt>
__global__ void matrix_matrix_add_kernel(TElem *a, TElem *b, TInt nRow, TInt nCol)
{
	if (blockIdx.x < nRow && threadIdx.x < nCol) {
		TInt i = blockDim.x * blockIdx.x + threadIdx.x;
		a[i] += b[i];
	}
}

template <typename TElem, typename TInt>
__global__ void matrix_matrix_sub_kernel(TElem *a, TElem *b, TInt nRow, TInt nCol)
{
	if (blockIdx.x < nRow && threadIdx.x < nCol) {
		TInt i = blockDim.x * blockIdx.x + threadIdx.x;
		a[i] -= b[i];
	}
}

// future : optimization
template <typename TElem, typename TInt>
__global__ void matrix_matrix_mult_kernel(TElem *a, TElem *b, TElem *c, TInt nRow0, TInt nCol0, TInt nCol1)
{
	TInt col = blockDim.x*blockIdx.x + threadIdx.x;
	TInt row = blockDim.y*blockIdx.y + threadIdx.y;

	TElem sum=0;
	if (col < nCol1 && row < nRow0) {
		for (TInt k=0; k<nCol0; k++) {
			sum += a[row*nCol0+ k] * b[k*nCol1 + col];
		}
	}
	c[row*nCol1+col] = sum;
}

template <typename TElem, typename TInt>
void matrix_element_add(TElem *a, TElem b, TInt nRow, TInt nCol)
{
	matrix_element_add_kernel<TElem, TInt> <<<nRow, nCol>>>(a, b, nRow, nCol);
}

template <typename TElem, typename TInt>
void matrix_element_sub(TElem *a, TElem b, TInt nRow, TInt nCol)
{
	matrix_element_sub_kernel<TElem, TInt> <<<nRow, nCol>>>(a, b, nRow, nCol);
}

template <typename TElem, typename TInt>
void matrix_element_mult(TElem *a, TElem b, TInt nRow, TInt nCol)
{
	matrix_element_mult_kernel<TElem, TInt> <<<nRow, nCol>>>(a, b, nRow, nCol);
}

template <typename TElem, typename TInt>
void matrix_element_div(TElem *a, TElem b, TInt nRow, TInt nCol)
{
	matrix_element_div_kernel<TElem, TInt> <<<nRow, nCol>>>(a, b, nRow, nCol);
}

template <typename TElem, typename TInt>
void matrix_matrix_add(TElem *a, TElem *b, TInt nRow, TInt nCol)
{
	matrix_matrix_add_kernel<TElem, TInt> <<<nRow, nCol>>>(a, b, nRow, nCol);
}

template <typename TElem, typename TInt>
void matrix_matrix_sub(TElem *a, TElem *b, TInt nRow, TInt nCol)
{
	matrix_matrix_sub_kernel<TElem, TInt> <<<nRow, nCol>>>(a, b, nRow, nCol);
}

// future : optimization
template <typename TElem, typename TInt>
void matrix_matrix_mult(TElem *a, TElem *b, TElem *c, TInt nRow0, TInt nCol0, TInt nCol1)
{
	dim3 dimGrid(1,1);
	dim3 dimBlock(nCol1,nRow0);
	matrix_matrix_mult_kernel<TElem, TInt> 
		<<<dimGrid, dimBlock>>> (a, b, c, nRow0, nCol0, nCol1);
}

}

#include "cuspCUDA.tpp"
