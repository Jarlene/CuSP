#include "hip/hip_runtime.h"
#include "cuspCUDA.cuh"
#include "cuspCUDA.hpp"

namespace cusp {

template <typename TElem, typename TInt>
__global__ void matrix_element_add_kernel(TElem *a, TElem b, TInt row, TInt col)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (blockIdx.x < row && threadIdx.x < col)
		a[i] += b;
}

template <typename TElem, typename TInt>
void matrix_element_add(TElem *a, TElem b, TInt row, TInt col)
{
	matrix_element_add_kernel<TElem, TInt> <<<row, col>>>(a, b, row, col);
}

template void matrix_element_add<float, unsigned int> 
	(float *a, float b, unsigned int row, unsigned  int col);
template void matrix_element_add<float, unsigned long long> 
	(float *a, float b, unsigned long long row, unsigned long long col);
template void matrix_element_add<double, unsigned int> 
	(double *a, double b, unsigned int row, unsigned  int col);
template void matrix_element_add<double, unsigned long long> 
	(double *a, double b, unsigned long long row, unsigned long long col);

}
