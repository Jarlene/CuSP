#include "hip/hip_runtime.h"
#include "cuspCUDA.cuh"
#include "cuspCUDA.hpp"

namespace cusp {

template <typename TElem, typename TInt>
__global__ void matrix_element_add_kernel(TElem *a, TElem b, TInt row, TInt col)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (blockIdx.x < row && threadIdx.x < col)
		a[i] += b;
}

template <typename TElem, typename TInt>
void matrix_element_add(TElem *a, TElem b, TInt row, TInt col)
{
	matrix_element_add_kernel<TElem, TInt> <<<row, col>>>(a, b, row, col);
}

template void matrix_element_add<char, int> 
	(char *a, char b, int row, int col);
template void matrix_element_add<char, unsigned int> 
	(char *a, char b, unsigned int row, unsigned  int col);
template void matrix_element_add<char, unsigned long long> 
	(char *a, char b, unsigned long long row, unsigned long long col);
template void matrix_element_add<short, int> 
	(short *a, short b, int row, int col);
template void matrix_element_add<short, unsigned int> 
	(short *a, short b, unsigned int row, unsigned  int col);
template void matrix_element_add<short, unsigned long long> 
	(short *a, short b, unsigned long long row, unsigned long long col);
template void matrix_element_add<int, int> 
	(int *a, int b, int row, int col);
template void matrix_element_add<int, unsigned int> 
	(int *a, int b, unsigned int row, unsigned  int col);
template void matrix_element_add<int, unsigned long long> 
	(int *a, int b, unsigned long long row, unsigned long long col);
template void matrix_element_add<float, int> 
	(float *a, float b, int row, int col);
template void matrix_element_add<float, unsigned int> 
	(float *a, float b, unsigned int row, unsigned  int col);
template void matrix_element_add<float, unsigned long long> 
	(float *a, float b, unsigned long long row, unsigned long long col);
template void matrix_element_add<double, int> 
	(double *a, double b, int row, int col);
template void matrix_element_add<double, unsigned int> 
	(double *a, double b, unsigned int row, unsigned  int col);
template void matrix_element_add<double, unsigned long long> 
	(double *a, double b, unsigned long long row, unsigned long long col);

}
