#include "hip/hip_runtime.h"
#include "array.cuh"

namespace cusp {

template <typename TElem, typename TInt>
__global__
void assignWithAdd(TElem *a, const TElem b)
{
	int i = threadIdx.x;
	a[i] += b;
}

}
